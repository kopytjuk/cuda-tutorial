#include <stdio.h>
#include "kernels.cu"

int main(void) {

    int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    printf("Using %s.\n\n", props.name);

    int a, b, c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    a = 40;
    b = 2;

    printf("Calculating %d+%d on GPU...\n", a, b);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("%d + %d = %d!\n", a, b, c);

    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}